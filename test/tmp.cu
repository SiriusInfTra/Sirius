
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

#define CUDA_CALL(func) do { \
    auto error = func; \
    if (error != hipSuccess) { \
    std::cout << #func << " " << hipGetErrorString(error); \
      exit(EXIT_FAILURE); \
    } \
  } while (0);

__global__ void kernel(int n, int *x) {
  for (int i = 0; i < n; i++) {
    x[i] = x[i] + 1;
  }
}

int main() {
  int n = 1 << 20;
  int *x, *y;
  CUDA_CALL(hipHostMalloc(&y, n * sizeof(int), hipHostMallocDefault));
  CUDA_CALL(hipMalloc(&x, 2 * n * sizeof(int)));
  for (int i = 0; i < n; i++) {
    y[i] = i;
  }

  hipStream_t s1, s2;
  CUDA_CALL(hipStreamCreate(&s1));
  CUDA_CALL(hipStreamCreate(&s2));

  for (int i = 0; i < 10; i++) {
    CUDA_CALL(hipMemcpyAsync(x, y, 2  * n * sizeof(int), hipMemcpyHostToDevice, s2));
    kernel<<<1, 1, 0, s1>>>(n, x);
  }
  CUDA_CALL(hipStreamSynchronize(s1));
  CUDA_CALL(hipStreamSynchronize(s2));
  
  int *z;
  hipHostMalloc(&z, n * sizeof(int), hipHostMallocDefault);
  CUDA_CALL(hipMemcpyAsync(z, x, n * sizeof(int), hipMemcpyDeviceToHost, s1));
  for (int i = 0; i < 20; i++) {
    std::cout << z[i] << " ";
  }
  std::cout << std::endl;
  cout << "Success!" << endl;
  return 0;
}